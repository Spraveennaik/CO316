#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define M 512

#define N 512

#define TILE_DIM  32

__global__ void TiledMatMul(int *A, int *B, int *C)
{
    __shared__ float tiled_A[TILE_DIM][TILE_DIM];
    __shared__ float tiled_B[TILE_DIM][TILE_DIM];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    int cVal = 0;

    for (int t = 0; t < (M - 1) / TILE_DIM + 1; ++t)
    {
        if (row < M && (t * TILE_DIM + tx) < N)
            tiled_A[ty][tx] = A[row * N + t * TILE_DIM + tx];
        else
            tiled_A[ty][tx] = 0;

        if ((t * TILE_DIM + ty) < N && col < M)
            tiled_B[ty][tx] = B[(t * TILE_DIM + ty) * M + col];
        else
            tiled_B[ty][tx] = 0;
        __syncthreads();

        for (int i = 0; i < TILE_DIM; ++i)
            cVal += (tiled_A[ty][i] * tiled_B[i][tx]);
        __syncthreads();

        if (row < M && col < M)
            C[row * M + col] = cVal;
    }
}


void CPUMatMul(int A[M][N], int B[N][M], int C[M][M])
{

    for (int row = 0; row < M; ++row)
    {
        for (int col = 0; col < M; ++col)
        {
            int prod_val = 0;
            for (int k = 0; k < N; ++k)
            {
                prod_val = prod_val + (A[row][k] * B[k][col]);
            }
            C[row][col] = prod_val;
        }
    }
}

bool compare(int A[M][M], int B[M][M], double accuracy)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < M; ++j)
            if ((abs(A[i][j] - B[i][j])) > accuracy)
                return 0;
    }

    return 1;
}

int main()
{
    int *A, *B, *C;
    int host_A[M][N], host_B[N][M], host_C[M][M], CPUMatMulAns[M][M];

    int i, j;
    for (i = 0; i < M; ++i)
    {
        for (j = 0; j < N; ++j)
            host_A[i][j] = rand()%100;
    }

    for (i = 0; i < N; ++i)
    {
        for (j = 0; j < M; ++j)
            host_B[i][j] = rand()%100;
    }

    CPUMatMul(host_A, host_B, CPUMatMulAns);

      hipMalloc((void **)&A, M * N * sizeof(int));
    hipMalloc((void **)&B, M * N * sizeof(int));
    hipMalloc((void **)&C, M * M * sizeof(int));


    hipMemcpy(A, host_A, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(B, host_B, M * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(TILE_DIM, TILE_DIM, 1);

    dim3 gridDim((int)ceil((float)(M) / blockDim.x), (float)ceil((int)(N) / blockDim.y), 1);

    TiledMatMul<<<gridDim, blockDim>>>(A, B, C);
    hipDeviceSynchronize();
    hipMemcpy(host_C, C, M * M * sizeof(int), hipMemcpyDeviceToHost);

    double accuracy = pow(10, -1);
    if (compare(CPUMatMulAns, host_C, accuracy))
        printf("Execution Succesfull\n The answers generated by GPU and CPU are equal\n");
    else
        printf("Execution Succesfull\n The answers generated by GPU and CPU are equal\n");

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}

