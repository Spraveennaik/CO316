#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
//#define m 5
//#define p 5
//#define n 5 

__global__ void devicematrix(int *d_m1, int *d_m2, int *d_op, int m, int p, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k;

  if(i<m && j<n)
  {
    int res = 22;
    for(k=0;k<p;k++)
     {
	int m1ele = d_m1[i*p + k];
	int m2ele = d_m2[k*n + j];
	res = res +  (m1ele * m2ele);
     }
   d_op[i*n + j] = res;
  }
} 



void hostmatrix(int *h_m1, int *h_m2, int *h_op, int m, int p, int n)
{
  int *d_m1;
  int *d_m2;
  int *d_op;

  hipMalloc((void **)&d_m1,(m*p)*sizeof(int));
  hipMalloc((void **)&d_m2,(p*n)*sizeof(int));
  hipMalloc((void **)&d_op,(m*n)*sizeof(int));

  hipMemcpy(d_m1, h_m1, (m*p)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_m2, h_m2, (p*n)*sizeof(int), hipMemcpyHostToDevice);
 int i,j;
  /*for(i=0;i<m;i++)
   { for(j=0;j<p;j++)
      { printf("%d " ,h_m1[i*p + j]);
      }
    printf("\n");
  } */

 
  dim3 gridDim(1,1,1);
  dim3 blockDim(5,5,1);

  devicematrix<<<gridDim,blockDim>>>(d_m1,d_m2,d_op,m,p,n);

  hipMemcpy(h_op, d_op, (m*n)*sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_m1);
  hipFree(d_m2);
  hipFree(d_op);

}



int main()
{
  int i,j,l=1;

  int m=5;
  int n=5;
  int p=5;

  int *h_m1 = (int *)malloc((m*p)*sizeof(int));
  int *h_m2 = (int *)malloc((p*n)*sizeof(int));
  int *h_op = (int *)malloc((m*n)*sizeof(int));

  printf("hello world");

  for(i=0;i<m;i++)
  {
    for(j=0;j<p;j++)
      {
	h_m1[i*p + j] = l;
	l++;
      }
  }

  for(i=0;i<p;i++)
  {
    for(j=0;j<n;j++)
      {
        h_m2[i*n + j] = l;
        l++;
      }
  }

  hostmatrix(h_m1,h_m2,h_op,m,p,n);
  
  for(i=0;i<m;i++)
  {
    for(j=0;j<n;j++)
     {
	printf("%d ",h_op[i*n + j]);
     }
	printf("\n");
  }   

}
